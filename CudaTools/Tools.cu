#include "hip/hip_runtime.h"
#include "Tools.h"



CudaTools::CudaTools()
{

}

CudaTools::~CudaTools()
{

}



__global__ void redact_areas(uchar3* imageIn, uchar3* imageOut, int width, int height, int4* rects, int num_rects, int block_size)
{
	// This function attempts to obscure (redact) areas of an image defined by the rects passed in, i.e. pixels inside the rects
	// are "redacted".  
	// The redaction process simply covers a specified rectangle with square blocks of solid color.  The size of these blocks
	// is set by the parameter block_size (in pixels).  The color of each block it taken from the center pixel of each block; every
	// pixel inside the block is set to this color.
	//
	// For example, given a rectangle of 10,10,50,80 (x,y,w,h format) and a block size of 8.  There will be at least a 5 x 9 blocks 
	// created to cover this region.  There may actually be a few more, as some padding around the edges is added.
	//
	// Parameters:
	//		image = the image data to work on
	//		width, height = the pixel dimension of the data
	//		rects = pointer to a array of rectangles (data packed into an int4 struct)
	//		num_rects = number of rectangles in the array above
	//		block_size = size of the blocks in pixels (typical would be 8 or 16)


	// calc x,y position of pixel to operate on
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside panel
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside panel

														// make sure we don't try to operate outside the image
	if (x >= width) return;
	if (y >= height) return;

	bool redact = false;

	// test to see if this pixel is inside one of the rects
	for (int i = 0; i < num_rects; i++)
	{
		int xr = rects[i].x;
		int yr = rects[i].y;
		int wr = rects[i].z;
		int hr = rects[i].w;

		if (x >= xr && y >= yr && x < (xr + wr) && y < (yr + hr))
		{
			redact = true;

			// redaction area origin
			int rao_x = xr - block_size / 2;
			int rao_y = yr - block_size / 2;
			if (rao_x < 0) rao_x = 0;
			if (rao_y < 0) rao_y = 0;

			// number of rows,cols of redaction blocks
			//int num_blocks_x = (wr + block_size) / (block_size)+1;
			//int num_blocks_y = (hr + block_size) / (block_size)+1;

			// redaction block row and col
			int rb_row = (y - rao_y) / block_size;
			int rb_col = (x - rao_x) / block_size;

			// redaction block center (this is where we get the color used for all pixels in this redaction block)
			int rbc_x = rao_x + (rb_col * block_size) + (block_size / 2);
			int rbc_y = rao_y + (rb_row * block_size) + (block_size / 2);
			if (rbc_x >= width) rbc_x = width - 1;
			if (rbc_y >= height) rbc_y = height - 1;

			// get the color to be set for this entire redaction block
			uchar3 color = imageIn[rbc_y * width + rbc_x];

			// set the color of this pixel
			imageOut[y * width + x] = color;

			// can't also be inside any other window, so break out of for loop
			break;
		}
	}

	if (!redact) // not inside any of the redaction rectangles, so just copy the pixel from imageIn to imageOut
	{
		imageOut[y*width + x] = imageIn[y*width + x];
	}
}


void CudaTools::RedactAreas(uchar3* rgb_image_in, uchar3* rgb_image_out, int width, int height, int4* rects, int num_rects, int block_size)
{
	dim3 block, grid;
	block.x = 32; block.y = 16; block.z = 1;

	grid.x = (width + block.x - 1) / block.x;
	grid.y = (height + block.y - 1) / block.y;
	grid.z = 1;

	redact_areas << <grid, block >> > (rgb_image_in, rgb_image_out, width, height, rects, num_rects, block_size);
}

